#include "hip/hip_runtime.h"


/*
 *  Global Switches
 */

#ifndef SAC_DO_CHECK
#define SAC_DO_CHECK                             0
#endif
#ifndef SAC_DO_CHECK_TYPE
#define SAC_DO_CHECK_TYPE                        0
#endif
#ifndef SAC_DO_CHECK_GPU
#define SAC_DO_CHECK_GPU                         0
#endif
#ifndef SAC_DO_CHECK_BOUNDARY
#define SAC_DO_CHECK_BOUNDARY                    0
#endif
#ifndef SAC_DO_CHECK_MALLOC
#define SAC_DO_CHECK_MALLOC                      0
#endif
#ifndef SAC_DO_CHECK_ERRNO
#define SAC_DO_CHECK_ERRNO                       0
#endif
#ifndef SAC_DO_CHECK_HEAP
#define SAC_DO_CHECK_HEAP                        0
#endif
#ifndef SAC_DO_CHECK_DISTMEM
#define SAC_DO_CHECK_DISTMEM                     0
#endif
#ifndef SAC_DO_CHECK_DISTMEMPHM
#define SAC_DO_CHECK_DISTMEMPHM                  0
#endif

#define SAC_DO_PHM                               0
#define SAC_DO_APS                               0
#define SAC_DO_DAO                               0
#define SAC_DO_MSCA                              0
#define SAC_DO_COMPILE_MODULE                    0

#ifndef SAC_DO_PROFILE
#define SAC_DO_PROFILE                           0
#endif
#ifndef SAC_DO_PROFILE_WITH
#define SAC_DO_PROFILE_WITH                      0
#endif
#ifndef SAC_DO_PROFILE_FUN
#define SAC_DO_PROFILE_FUN                       0
#endif
#ifndef SAC_DO_PROFILE_INL
#define SAC_DO_PROFILE_INL                       0
#endif
#ifndef SAC_DO_PROFILE_LIB
#define SAC_DO_PROFILE_LIB                       0
#endif
#ifndef SAC_DO_PROFILE_MEM
#define SAC_DO_PROFILE_MEM                       0
#endif
#ifndef SAC_DO_PROFILE_OPS
#define SAC_DO_PROFILE_OPS                       0
#endif
#ifndef SAC_DO_PROFILE_CUDA
#define SAC_DO_PROFILE_CUDA                      0
#endif
#ifndef SAC_DO_PROFILE_DISTMEM
#define SAC_DO_PROFILE_DISTMEM                   0
#endif

#ifndef SAC_DO_TRACE
#define SAC_DO_TRACE                             0
#endif
#ifndef SAC_DO_TRACE_REF
#define SAC_DO_TRACE_REF                         0
#endif
#ifndef SAC_DO_TRACE_MEM
#define SAC_DO_TRACE_MEM                         0
#endif
#ifndef SAC_DO_TRACE_PRF
#define SAC_DO_TRACE_PRF                         0
#endif
#ifndef SAC_DO_TRACE_FUN
#define SAC_DO_TRACE_FUN                         0
#endif
#ifndef SAC_DO_TRACE_WL
#define SAC_DO_TRACE_WL                          0
#endif
#ifndef SAC_DO_TRACE_AA
#define SAC_DO_TRACE_AA                          0
#endif
#ifndef SAC_DO_TRACE_MT
#define SAC_DO_TRACE_MT                          0
#endif
#ifndef SAC_DO_TRACE_GPU
#define SAC_DO_TRACE_GPU                         0
#endif
#ifndef SAC_DO_TRACE_RTSPEC
#define SAC_DO_TRACE_RTSPEC                      0
#endif
#ifndef SAC_DO_TRACE_DISTMEM
#define SAC_DO_TRACE_DISTMEM                     0
#endif

#ifndef SAC_DO_CACHESIM
#define SAC_DO_CACHESIM                          0
#endif
#ifndef SAC_DO_CACHESIM_ADV
#define SAC_DO_CACHESIM_ADV                      0
#endif
#ifndef SAC_DO_CACHESIM_GLOBAL
#define SAC_DO_CACHESIM_GLOBAL                   1
#endif
#ifndef SAC_DO_CACHESIM_FILE
#define SAC_DO_CACHESIM_FILE                     0
#endif
#ifndef SAC_DO_CACHESIM_PIPE
#define SAC_DO_CACHESIM_PIPE                     0
#endif
#ifndef SAC_DO_CACHESIM_IMDT
#define SAC_DO_CACHESIM_IMDT                     1
#endif

/*
 * Setup for Multi Threaded Data Parallelism
 */
#define SAC_DO_MULTITHREAD                       0
#define SAC_DO_THREADS_STATIC                    1
#define SAC_DO_MT_CREATE_JOIN                    0
#define SAC_DO_MT_PTHREAD                        0
#define SAC_DO_MT_LPEL                           0

/*
 * Setup for OMP Data Parallelism
 */
#define SAC_DO_MT_OMP                            0
#define SAC_DO_OMP_MACROS                        0

/*
 * Setup for MUTC
 */
#define SAC_MUTC_FUNAP_AS_CREATE                 0
#define SAC_MUTC_THREAD_MALLOC                   0
#define SAC_MUTC_DISABLE_THREAD_MEM              0
#define SAC_MUTC_BENCH                           0
#define SAC_MUTC_MACROS                          0
#define SAC_MUTC_RC_INDIRECT                     0
#define SAC_MUTC_SEQ_DATA_PARALLEL               0

/*
 * Setup for GPU Data Parallelism
 */
#define SAC_CUDA_MACROS                          1

/*
 * Setup for Distributed Memory Data Parallelism
 */
#define SAC_DO_DISTMEM                           0
#define SAC_DO_DISTMEM_GASNET                    0
#define SAC_DO_DISTMEM_GPI                       0
#define SAC_DO_DISTMEM_MPI                       0
#define SAC_DO_DISTMEM_ARMCI                     0
#define SAC_DO_DISTMEM_ALLOC_CACHE_OUTSIDE_DSM   0
#define SAC_DO_DISTMEM_PTR_DESC                  0
#define SAC_DO_DISTMEM_PTR_CACHE                 1

/*
 * Setup for Task Parallelism
 */
#define SAC_DO_FP                                0

/*
 * Debugging Support
 */
#ifndef SAC_DEBUG_RC
#define SAC_DEBUG_RC                             0
#endif

#define SAC_DO_CUDA_FORCE_INIT 1


/*
 *  Global Settings
 */

#define SAC_FORCE_DESC_SIZE -1

/*
 *  MUTC Backend Specific Settings
 */
#define SAC_MUTC_RC_PLACES  1
#define SAC_MUTC_FORCE_SPAWN_FLAGS

#define SAC_C_EXTERN           extern "C"



/*
 *  Global Settings
 */

#ifndef NULL
#  ifdef __cplusplus
#    define NULL         0
#  else
#    define NULL         (void*) 0
#  endif
#endif

#define SAC_SET_TMPDIR              "/tmp"
#define SAC_SET_INITIAL_MASTER_HEAPSIZE      1048576
#define SAC_SET_INITIAL_WORKER_HEAPSIZE      65536
#define SAC_SET_INITIAL_UNIFIED_HEAPSIZE     0

#ifndef SAC_SET_RTSPEC_THREADS
#define SAC_SET_RTSPEC_THREADS              1
#endif

#ifndef SAC_SET_MTMODE
#define SAC_SET_MTMODE               0
#endif

#define SAC_SET_CPU_BIND_STRATEGY 0
#define SAC_SET_BARRIER_TYPE               0
#define SAC_SET_SMART_DECISIONS            0
#define SAC_SET_SMART_FILENAME           "default"
#define SAC_SET_SMART_ARCH               "(null)"
#define SAC_SET_SMART_PERIOD               500
#ifndef SAC_SET_THREADS_MAX
#define SAC_SET_THREADS_MAX          128
#endif

#ifndef SAC_SET_THREADS
#define SAC_SET_THREADS              1
#endif

#ifndef SAC_OMP_ACTIVE_LEVEL
#define SAC_OMP_ACTIVE_LEVEL          1
#endif

#ifndef SAC_SET_MASTERCLASS
#define SAC_SET_MASTERCLASS          0
#endif

#define SAC_SET_NUM_SCHEDULERS       0

#define SAC_SET_CACHE_1_SIZE         -1
#define SAC_SET_CACHE_1_LINE         4
#define SAC_SET_CACHE_1_ASSOC        1
#define SAC_SET_CACHE_1_WRITEPOL     SAC_CS_default
#define SAC_SET_CACHE_1_MSCA_FACTOR  0.00

#define SAC_SET_CACHE_2_SIZE         -1
#define SAC_SET_CACHE_2_LINE         4
#define SAC_SET_CACHE_2_ASSOC        1
#define SAC_SET_CACHE_2_WRITEPOL     SAC_CS_default
#define SAC_SET_CACHE_2_MSCA_FACTOR  0.00

#define SAC_SET_CACHE_3_SIZE         -1
#define SAC_SET_CACHE_3_LINE         4
#define SAC_SET_CACHE_3_ASSOC        1
#define SAC_SET_CACHE_3_WRITEPOL     SAC_CS_default
#define SAC_SET_CACHE_3_MSCA_FACTOR  0.00

#define SAC_SET_CACHESIM_HOST        ""
#define SAC_SET_CACHESIM_FILE        "a.out.cs"
#define SAC_SET_CACHESIM_DIR         "/tmp"
#define SAC_SET_MAXFUN               0
#define SAC_SET_MAXFUNAP             1
#define SBLOCKSZ               16
#define LBLOCKSZ               256



/*
 *  Includes
 */


#include "sac.h"


#if SAC_OMP_MACROS

#include "omp.h"

#endif

#if SAC_CUDA_MACROS

#include <stdio.h>


#include <hip/hip_runtime.h>


#include <hip/hip_runtime.h>


#include <algorithm>

#endif

/*
 *  SAC-Program cuda.sac :
 */


/*
 *  type definitions
 */

SAC_ND_TYPEDEF ((SACt_sacprelude_d__SACarg, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SAC_hidden)

SAC_C_EXTERN SACt_sacprelude_d__SACarg SACARGcopy ( SACt_sacprelude_d__SACarg);
SAC_C_EXTERN void SACARGfree ( SACt_sacprelude_d__SACarg);



/*
 *  Global Definitions
 */

SAC_PF_DEFINE()
SAC_HM_DEFINE()


/*
 *  prototypes for locals (FUNDEFS)
 */

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__partitionSlicer__i_S__i_S__i_S__i_S__i_S, , 9, out, int, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), out, int, (SAC_arg_2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), out, int, (SAC_arg_3, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), out, int, (SAC_arg_4, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_min, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_max, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_axis, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_lb, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_ub, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__partitionSlicer__i_S__i_S__i_S__i_S__i_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_out ( (SAC_arg_2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_out ( (SAC_arg_3, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_out ( (SAC_arg_4, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_min, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_max, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_axis, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_lb, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_ub, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__partitionIntersectMax__i_S__i_S, , 3, out, int, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_PWLbound1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_ivmin, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__partitionIntersectMax__i_S__i_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_PWLbound1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_ivmin, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__isPartitionIntersectNull__i_S__i_S__i_S__i_S, , 5, out, bool, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idxmin, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idxmax, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_bound1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_bound2, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__isPartitionIntersectNull__i_S__i_S__i_S__i_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_idxmin, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_idxmax, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_bound1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_bound2, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selSxADistmemLocal__i_S__i_S, , 3, out, int, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selSxADistmemLocal__i_S__i_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selSxADistmemLocal__i_S__f_S, , 3, out, float, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, float, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selSxADistmemLocal__i_S__f_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selSxADistmemLocal__i_S__d_S, , 3, out, double, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, double, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selSxADistmemLocal__i_S__d_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selSxADistmemLocal__i_S__bl_S, , 3, out, bool, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, bool, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selSxADistmemLocal__i_S__bl_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selSxADistmemLocal__i_S__c_S, , 3, out, unsigned char, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, unsigned char, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selSxADistmemLocal__i_S__c_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selSxADistmemLocal__i_S__b_S, , 3, out, byte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, byte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selSxADistmemLocal__i_S__b_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selSxADistmemLocal__i_S__s_S, , 3, out, short, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, short, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selSxADistmemLocal__i_S__s_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selSxADistmemLocal__i_S__l_S, , 3, out, long, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, long, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selSxADistmemLocal__i_S__l_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selSxADistmemLocal__i_S__ll_S, , 3, out, longlong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, longlong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selSxADistmemLocal__i_S__ll_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selSxADistmemLocal__i_S__ub_S, , 3, out, ubyte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ubyte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selSxADistmemLocal__i_S__ub_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selSxADistmemLocal__i_S__us_S, , 3, out, ushort, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ushort, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selSxADistmemLocal__i_S__us_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selSxADistmemLocal__i_S__ui_S, , 3, out, uint, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, uint, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selSxADistmemLocal__i_S__ui_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selSxADistmemLocal__i_S__ul_S, , 3, out, ulong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selSxADistmemLocal__i_S__ul_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selSxADistmemLocal__i_S__ull_S, , 3, out, ulonglong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulonglong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selSxADistmemLocal__i_S__ull_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__partitionIntersectMin__i_S__i_S, , 3, out, int, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_PWLbound2, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_ivmax, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__partitionIntersectMin__i_S__i_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_PWLbound2, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_ivmax, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__adjustLacFunParams__bl_S__i_S__i_S, , 4, out, int, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, bool, (SACl_p, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_i, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_iv, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__adjustLacFunParams__bl_S__i_S__i_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_p, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_i, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_iv, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf__MAIN__main, , 1, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf__MAIN__main, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__prod__i_S, , 2, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_v, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__prod__i_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_v, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__adjustLacFunParamsReshape__bl_S__i_S__i_S__i_S, , 5, out, int, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, bool, (SACl_p, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_i, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_iv, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_shp, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__adjustLacFunParamsReshape__bl_S__i_S__i_S__i_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_p, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_i, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_iv, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_shp, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__eq__i_S__i_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__eq__i_S__i_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__eq__f_S__f_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, float, (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, float, (SACl_B, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__eq__f_S__f_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__eq__d_S__d_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, double, (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), in, double, (SACl_B, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__eq__d_S__d_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__eq__bl_S__bl_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, bool, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, bool, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__eq__bl_S__bl_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__eq__c_S__c_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, unsigned char, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in, unsigned char, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__eq__c_S__c_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__eq__b_S__b_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, byte, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), in, byte, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__eq__b_S__b_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__eq__s_S__s_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, short, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), in, short, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__eq__s_S__s_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__eq__l_S__l_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, long, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), in, long, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__eq__l_S__l_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__eq__ll_S__ll_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, longlong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), in, longlong, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__eq__ll_S__ll_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__eq__ub_S__ub_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, ubyte, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), in, ubyte, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__eq__ub_S__ub_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__eq__us_S__us_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, ushort, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), in, ushort, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__eq__us_S__us_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__eq__ui_S__ui_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, uint, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), in, uint, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__eq__ui_S__ui_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__eq__ul_S__ul_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, ulong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), in, ulong, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__eq__ul_S__ul_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__eq__ull_S__ull_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, ulonglong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), in, ulonglong, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__eq__ull_S__ull_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__gridFiller__i_S__i_S__i_S__i_S__i_S, , 8, out, int, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), out, int, (SAC_arg_2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), out, int, (SAC_arg_3, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_lb, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_ub, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_wdth, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_dim, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_maxwidth, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__gridFiller__i_S__i_S__i_S__i_S__i_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_out ( (SAC_arg_2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_out ( (SAC_arg_3, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_lb, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_ub, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_wdth, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_dim, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_maxwidth, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__isPartitionIntersect1Part__i_S__i_S__i_S__i_S, , 5, out, bool, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idxmin, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idxmax, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_bound1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_bound2, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__isPartitionIntersect1Part__i_S__i_S__i_S__i_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_idxmin, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_idxmax, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_bound1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_bound2, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__zero__i_S, , 2, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__zero__i_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__zero__d_S, , 2, out, double, (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), in, double, (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__zero__d_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__zero__f_S, , 2, out, float, (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, float, (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__zero__f_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__zero__c_S, , 2, out, unsigned char, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in, unsigned char, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__zero__c_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__zero__bl_S, , 2, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, bool, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__zero__bl_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__zero__b_S, , 2, out, byte, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), in, byte, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__zero__b_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__zero__s_S, , 2, out, short, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), in, short, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__zero__s_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__zero__l_S, , 2, out, long, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), in, long, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__zero__l_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__zero__ll_S, , 2, out, longlong, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), in, longlong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__zero__ll_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__zero__ub_S, , 2, out, ubyte, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), in, ubyte, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__zero__ub_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__zero__us_S, , 2, out, ushort, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), in, ushort, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__zero__us_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__zero__ui_S, , 2, out, uint, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), in, uint, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__zero__ui_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__zero__ul_S, , 2, out, ulong, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), in, ulong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__zero__ul_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__zero__ull_S, , 2, out, ulonglong, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), in, ulonglong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__zero__ull_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__partitionMax__i_S__i_S, , 3, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_x, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_y, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__partitionMax__i_S__i_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_x, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_y, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__partitionMin__i_S__i_S, , 3, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_x, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_y, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__partitionMin__i_S__i_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_x, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_y, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___PL_PL__i_S, , 2, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_a, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___PL_PL__i_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_a, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__sel__i_S__i_S, , 3, out, int, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__sel__i_S__i_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__sel__i_S__f_S, , 3, out, float, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, float, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__sel__i_S__f_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__sel__i_S__d_S, , 3, out, double, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, double, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__sel__i_S__d_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__sel__i_S__bl_S, , 3, out, bool, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, bool, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__sel__i_S__bl_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__sel__i_S__c_S, , 3, out, unsigned char, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, unsigned char, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__sel__i_S__c_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__sel__i_S__b_S, , 3, out, byte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, byte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__sel__i_S__b_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__sel__i_S__s_S, , 3, out, short, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, short, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__sel__i_S__s_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__sel__i_S__l_S, , 3, out, long, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, long, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__sel__i_S__l_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__sel__i_S__ll_S, , 3, out, longlong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, longlong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__sel__i_S__ll_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__sel__i_S__ub_S, , 3, out, ubyte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ubyte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__sel__i_S__ub_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__sel__i_S__us_S, , 3, out, ushort, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ushort, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__sel__i_S__us_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__sel__i_S__ui_S, , 3, out, uint, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, uint, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__sel__i_S__ui_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__sel__i_S__ul_S, , 3, out, ulong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__sel__i_S__ul_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d__sel__i_S__ull_S, , 3, out, ulonglong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulonglong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d__sel__i_S__ull_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selVxADistmemLocal__i_S__i_S, , 3, out, int, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selVxADistmemLocal__i_S__i_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selVxADistmemLocal__i_S__f_S, , 3, out, float, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, float, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selVxADistmemLocal__i_S__f_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selVxADistmemLocal__i_S__d_S, , 3, out, double, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, double, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selVxADistmemLocal__i_S__d_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selVxADistmemLocal__i_S__bl_S, , 3, out, bool, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, bool, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selVxADistmemLocal__i_S__bl_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selVxADistmemLocal__i_S__c_S, , 3, out, unsigned char, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, unsigned char, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selVxADistmemLocal__i_S__c_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selVxADistmemLocal__i_S__b_S, , 3, out, byte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, byte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selVxADistmemLocal__i_S__b_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selVxADistmemLocal__i_S__s_S, , 3, out, short, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, short, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selVxADistmemLocal__i_S__s_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selVxADistmemLocal__i_S__l_S, , 3, out, long, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, long, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selVxADistmemLocal__i_S__l_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selVxADistmemLocal__i_S__ll_S, , 3, out, longlong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, longlong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selVxADistmemLocal__i_S__ll_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selVxADistmemLocal__i_S__ub_S, , 3, out, ubyte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ubyte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selVxADistmemLocal__i_S__ub_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selVxADistmemLocal__i_S__us_S, , 3, out, ushort, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ushort, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selVxADistmemLocal__i_S__us_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selVxADistmemLocal__i_S__ui_S, , 3, out, uint, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, uint, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selVxADistmemLocal__i_S__ui_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selVxADistmemLocal__i_S__ul_S, , 3, out, ulong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selVxADistmemLocal__i_S__ul_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACwf_sacprelude_d___selVxADistmemLocal__i_S__ull_S, , 3, out, ulonglong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulonglong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACwf_sacprelude_d___selVxADistmemLocal__i_S__ull_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong), SAC_ND_PARAM_in ( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf__MAIN__hiding, , 1, out, int, (SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf__MAIN__hiding, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf__MAIN__main, , 1, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf__MAIN__main, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__sel__i_X__ull_S, , 3, out, ulonglong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulonglong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__sel__i_X__ull_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__sel__i_X__ul_S, , 3, out, ulong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__sel__i_X__ul_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__sel__i_X__ui_S, , 3, out, uint, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, uint, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__sel__i_X__ui_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__sel__i_X__us_S, , 3, out, ushort, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ushort, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__sel__i_X__us_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__sel__i_X__ub_S, , 3, out, ubyte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ubyte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__sel__i_X__ub_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__sel__i_X__ll_S, , 3, out, longlong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, longlong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__sel__i_X__ll_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__sel__i_X__l_S, , 3, out, long, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, long, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__sel__i_X__l_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__sel__i_X__s_S, , 3, out, short, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, short, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__sel__i_X__s_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__sel__i_X__b_S, , 3, out, byte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, byte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__sel__i_X__b_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__sel__i_X__c_S, , 3, out, unsigned char, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, unsigned char, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__sel__i_X__c_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__sel__i_X__bl_S, , 3, out, bool, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, bool, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__sel__i_X__bl_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__sel__i_X__d_S, , 3, out, double, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, double, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__sel__i_X__d_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__sel__i_X__f_S, , 3, out, float, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, float, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__sel__i_X__f_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__sel__i_X__i_S, , 3, out, int, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__sel__i_X__i_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_1__ull_S, , 3, out, ulonglong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulonglong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_1__ull_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong), SAC_ND_PARAM_in ( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_X__ull_S, , 3, out, ulonglong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulonglong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_X__ull_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_1__ul_S, , 3, out, ulong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_1__ul_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong), SAC_ND_PARAM_in ( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_X__ul_S, , 3, out, ulong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_X__ul_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_1__ui_S, , 3, out, uint, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, uint, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_1__ui_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint), SAC_ND_PARAM_in ( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_X__ui_S, , 3, out, uint, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, uint, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_X__ui_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_1__us_S, , 3, out, ushort, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ushort, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_1__us_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort), SAC_ND_PARAM_in ( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_X__us_S, , 3, out, ushort, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ushort, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_X__us_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_1__ub_S, , 3, out, ubyte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ubyte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_1__ub_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte), SAC_ND_PARAM_in ( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_X__ub_S, , 3, out, ubyte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ubyte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_X__ub_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_1__ll_S, , 3, out, longlong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, longlong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_1__ll_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong), SAC_ND_PARAM_in ( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_X__ll_S, , 3, out, longlong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, longlong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_X__ll_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_1__l_S, , 3, out, long, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, long, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_1__l_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long), SAC_ND_PARAM_in ( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_X__l_S, , 3, out, long, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, long, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_X__l_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_1__s_S, , 3, out, short, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, short, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_1__s_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short), SAC_ND_PARAM_in ( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_X__s_S, , 3, out, short, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, short, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_X__s_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_1__b_S, , 3, out, byte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, byte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_1__b_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte), SAC_ND_PARAM_in ( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_X__b_S, , 3, out, byte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, byte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_X__b_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_1__c_S, , 3, out, unsigned char, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, unsigned char, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_1__c_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char), SAC_ND_PARAM_in ( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_X__c_S, , 3, out, unsigned char, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, unsigned char, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_X__c_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_1__bl_S, , 3, out, bool, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, bool, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_1__bl_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_X__bl_S, , 3, out, bool, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, bool, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_X__bl_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_1__d_S, , 3, out, double, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, double, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_1__d_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double), SAC_ND_PARAM_in ( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_X__d_S, , 3, out, double, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, double, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_X__d_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_1__f_S, , 3, out, float, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, float, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_1__f_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in ( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_X__f_S, , 3, out, float, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, float, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_X__f_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_1__i_S, , 3, out, int, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_1__i_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selVxADistmemLocal__i_X__i_S, , 3, out, int, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selVxADistmemLocal__i_X__i_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selSxADistmemLocal__i__ull_S, , 3, out, ulonglong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulonglong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selSxADistmemLocal__i__ull_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong), SAC_ND_PARAM_in ( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selSxADistmemLocal__i__ul_S, , 3, out, ulong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selSxADistmemLocal__i__ul_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong), SAC_ND_PARAM_in ( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selSxADistmemLocal__i__ui_S, , 3, out, uint, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, uint, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selSxADistmemLocal__i__ui_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint), SAC_ND_PARAM_in ( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selSxADistmemLocal__i__us_S, , 3, out, ushort, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ushort, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selSxADistmemLocal__i__us_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort), SAC_ND_PARAM_in ( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selSxADistmemLocal__i__ub_S, , 3, out, ubyte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ubyte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selSxADistmemLocal__i__ub_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte), SAC_ND_PARAM_in ( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selSxADistmemLocal__i__ll_S, , 3, out, longlong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, longlong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selSxADistmemLocal__i__ll_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong), SAC_ND_PARAM_in ( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selSxADistmemLocal__i__l_S, , 3, out, long, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, long, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selSxADistmemLocal__i__l_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long), SAC_ND_PARAM_in ( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selSxADistmemLocal__i__s_S, , 3, out, short, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, short, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selSxADistmemLocal__i__s_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short), SAC_ND_PARAM_in ( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selSxADistmemLocal__i__b_S, , 3, out, byte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, byte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selSxADistmemLocal__i__b_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte), SAC_ND_PARAM_in ( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selSxADistmemLocal__i__c_S, , 3, out, unsigned char, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, unsigned char, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selSxADistmemLocal__i__c_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char), SAC_ND_PARAM_in ( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selSxADistmemLocal__i__bl_S, , 3, out, bool, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, bool, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selSxADistmemLocal__i__bl_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selSxADistmemLocal__i__d_S, , 3, out, double, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, double, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selSxADistmemLocal__i__d_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double), SAC_ND_PARAM_in ( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selSxADistmemLocal__i__f_S, , 3, out, float, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, float, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selSxADistmemLocal__i__f_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in ( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___selSxADistmemLocal__i__i_S, , 3, out, int, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___selSxADistmemLocal__i__i_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__zero__ull_S, , 2, out, ulonglong, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), in, ulonglong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__zero__ull_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__zero__ul_S, , 2, out, ulong, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), in, ulong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__zero__ul_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__zero__ui_S, , 2, out, uint, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), in, uint, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__zero__ui_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__zero__us_S, , 2, out, ushort, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), in, ushort, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__zero__us_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__zero__ub_S, , 2, out, ubyte, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), in, ubyte, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__zero__ub_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__zero__ll_S, , 2, out, longlong, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), in, longlong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__zero__ll_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__zero__l_S, , 2, out, long, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), in, long, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__zero__l_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__zero__s_S, , 2, out, short, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), in, short, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__zero__s_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__zero__b_S, , 2, out, byte, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), in, byte, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__zero__b_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__zero__bl_S, , 2, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, bool, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__zero__bl_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__zero__c_S, , 2, out, unsigned char, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in, unsigned char, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__zero__c_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__zero__f_S, , 2, out, float, (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, float, (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__zero__f_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__zero__d_S, , 2, out, double, (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), in, double, (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__zero__d_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__zero__i_S, , 2, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__zero__i_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__eq__ull_S__ull_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, ulonglong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), in, ulonglong, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__eq__ull_S__ull_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__eq__ul_S__ul_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, ulong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), in, ulong, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__eq__ul_S__ul_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__eq__ui_S__ui_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, uint, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), in, uint, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__eq__ui_S__ui_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__eq__us_S__us_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, ushort, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), in, ushort, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__eq__us_S__us_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__eq__ub_S__ub_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, ubyte, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), in, ubyte, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__eq__ub_S__ub_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__eq__ll_S__ll_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, longlong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), in, longlong, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__eq__ll_S__ll_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__eq__l_S__l_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, long, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), in, long, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__eq__l_S__l_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__eq__s_S__s_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, short, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), in, short, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__eq__s_S__s_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__eq__b_S__b_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, byte, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), in, byte, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__eq__b_S__b_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__eq__c_S__c_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, unsigned char, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in, unsigned char, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__eq__c_S__c_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__eq__bl_S__bl_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, bool, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, bool, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__eq__bl_S__bl_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__eq__d_S__d_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, double, (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), in, double, (SACl_B, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__eq__d_S__d_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__eq__f_S__f_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, float, (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, float, (SACl_B, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__eq__f_S__f_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__eq__i_S__i_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__eq__i_S__i_S, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__adjustLacFunParams__bl_S__i_S__i_X, , 4, out, int, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, bool, (SACl_p, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_i, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_iv, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__adjustLacFunParams__bl_S__i_S__i_X, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_p, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_i, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_iv, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__adjustLacFunParamsReshape__bl_S__i_S__i_X__i_X, , 5, out, int, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, bool, (SACl_p, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_i, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_iv, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_shp, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__adjustLacFunParamsReshape__bl_S__i_S__i_X__i_X, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_p, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_i, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_iv, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_shp, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__isPartitionIntersectNull__i_X__i_X__i_X__i_X, , 5, out, bool, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idxmin, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idxmax, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_bound1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_bound2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__isPartitionIntersectNull__i_X__i_X__i_X__i_X, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_idxmin, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_idxmax, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_bound1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_bound2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__isPartitionIntersectNull__i__i__i_X__i_X, , 5, out, bool, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idxmin, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idxmax, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_bound1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_bound2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__isPartitionIntersectNull__i__i__i_X__i_X, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_idxmin, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_idxmax, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_bound1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_bound2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__partitionMin__i__i, , 3, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_x, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_y, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__partitionMin__i__i, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_x, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_y, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__partitionMax__i__i, , 3, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_x, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_y, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__partitionMax__i__i, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_x, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_y, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__isPartitionIntersect1Part__i_X__i_X__i_X__i_X, , 5, out, bool, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idxmin, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idxmax, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_bound1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_bound2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__isPartitionIntersect1Part__i_X__i_X__i_X__i_X, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_idxmin, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_idxmax, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_bound1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_bound2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__isPartitionIntersect1Part__i__i__i_X__i_X, , 5, out, bool, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idxmin, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idxmax, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_bound1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_bound2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__isPartitionIntersect1Part__i__i__i_X__i_X, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_idxmin, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_idxmax, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_bound1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_bound2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__partitionIntersectMax__i_X__i_X, , 3, out, int, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_PWLbound1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_ivmin, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__partitionIntersectMax__i_X__i_X, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_PWLbound1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_ivmin, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__partitionIntersectMax__i_X__i, , 3, out, int, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_PWLbound1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_ivmin, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__partitionIntersectMax__i_X__i, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_PWLbound1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_ivmin, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__partitionIntersectMin__i_X__i_X, , 3, out, int, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_PWLbound2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_ivmax, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__partitionIntersectMin__i_X__i_X, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_PWLbound2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_ivmax, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__partitionIntersectMin__i_X__i, , 3, out, int, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_PWLbound2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_ivmax, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__partitionIntersectMin__i_X__i, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_PWLbound2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_ivmax, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__prod__i_X, , 2, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_v, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__prod__i_X, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_v, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__partitionSlicer__i_X__i_X__i__i_X__i_X, , 9, out, int, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), out, int, (SAC_arg_2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), out, int, (SAC_arg_3, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), out, int, (SAC_arg_4, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_min, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_max, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_axis, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_lb, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_ub, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__partitionSlicer__i_X__i_X__i__i_X__i_X, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_out ( (SAC_arg_2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_out ( (SAC_arg_3, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_out ( (SAC_arg_4, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_min, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_max, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_axis, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_lb, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_ub, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__gridFiller__i_X__i_X__i_X__i__i_X, , 8, out, int, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), out, int, (SAC_arg_2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), out, int, (SAC_arg_3, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_lb, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_ub, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_wdth, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_dim, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_maxwidth, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__gridFiller__i_X__i_X__i_X__i__i_X, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_out ( (SAC_arg_2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_out ( (SAC_arg_3, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_lb, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_ub, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_wdth, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_dim, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_maxwidth, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___PL_PL__i, , 2, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_a, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___PL_PL__i, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_a, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d___ST__i__i, , 3, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_a, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d___ST__i__i, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_a, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in ( (SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN
/*
 * ND_FUN_DECL( SACf_sacprelude_d__and__bl__bl, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, bool, (SACl_a, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, bool, (SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DECL_FUN2 ( SACf_sacprelude_d__and__bl__bl, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_a, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in ( (SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool));

SAC_C_EXTERN
/*
 * CUDA_GLOBALFUN_DECL( SACf__MAIN___cuknl_5648_CUDA__i__i__i__i__i__i__i__id_100_100_100, 8, inout, int, (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 3, in, int, (SACp_ub_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_ub_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_ub_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_emal_4727__flat_13, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0)
 */
__global__ void SACf__MAIN___cuknl_5648_CUDA__i__i__i__i__i__i__i__id_100_100_100 (SAC_CUDA_PARAM_inout ( (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), int), int SAC_ND_A_MIRROR_SHAPE ((SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 0), int SAC_ND_A_MIRROR_SHAPE ((SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 1), int SAC_ND_A_MIRROR_SHAPE ((SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 2), int SAC_ND_A_MIRROR_SIZE ((SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, ))))))))))), int SAC_ND_A_MIRROR_DIM ((SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, ))))))))))), SAC_CUDA_PARAM_in ( (SACp_ub_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in ( (SACp_ub_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in ( (SACp_ub_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in ( (SACp_lb_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in ( (SACp_lb_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in ( (SACp_lb_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in ( (SACp_emal_4727__flat_13, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));
;



/*
 *  function definitions (FUNDEFS)
 */



/****************************************************************************
 * Wrapper function:
 * WITH-loop Count: 0
 * _MAIN::SACwf__MAIN__main(...) [ wrapper ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACwf__MAIN__main, , 1, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DEF_FUN_BEGIN2 ( SACwf__MAIN__main, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int)) {
	SAC_HM_DEFINE_THREAD_STATUS ( SAC_HM_single_threaded)
	SAC_MT_DEFINE_ST_SELF()

	{
		/*
		 * ND_DECL( (SACp_cwc_25, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
		 */
		SAC_ND_DECL__DATA ( (SACp_cwc_25, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
		SAC_ND_DECL__DESC ( (SACp_cwc_25, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
		SAC_NOTHING()


		SAC_INIT_LOCAL_MEM()
		/*
		 * ND_FUN_AP( SACf__MAIN__main, , 1, out, int, SAC_SET_NT_USG( FAG, (SACp_cwc_25, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))))
		 */
		SAC_ND_FUNAP2 ( SACf__MAIN__main,  SAC_ND_ARG_out ( SAC_SET_NT_USG ( FAG, (SACp_cwc_25, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int))

		/*
		 * ND_REFRESH__MIRROR( (SACp_cwc_25, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
		 */
		SAC_NOOP()

		/*
		 * ND_FUN_RET( , 1, out, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), (SACp_cwc_25, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
		 */
		SAC_ND_RET_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), (SACp_cwc_25, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
		return;
		SAC_CLEANUP_LOCAL_MEM()
	}
	/*
	   * ND_FUN_DEF_END( SACwf__MAIN__main, , 1, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
	   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * WITH-loop Count: 0
 * _MAIN::SACf__MAIN__hiding(...) [ body ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACf__MAIN__hiding, , 1, out, int, (SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DEF_FUN_BEGIN2 ( SACf__MAIN__hiding, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int)) {
	SAC_HM_DEFINE_THREAD_STATUS ( SAC_HM_single_threaded)
	SAC_MT_DEFINE_ST_SELF()

	{
		SAC_ND_DECL_CONST__DATA ((SACp_emal_4727__flat_13, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 42)
		SAC_ND_DECL_CONST__DATA ((SACp_emal_4726__cnstass_4710_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
		SAC_ND_DECL_CONST__DATA ((SACp_emal_4725__cnstass_4709_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
		SAC_ND_DECL_CONST__DATA ((SACp_emal_4724__cnstass_4708_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
		SAC_ND_DECL_CONST__DATA ((SACp_emal_4723__cnstass_4707_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 100)
		SAC_ND_DECL_CONST__DATA ((SACp_emal_4722__cnstass_4706_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 100)
		SAC_ND_DECL_CONST__DATA ((SACp_emal_4721__cnstass_4705_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 100)
		/*
		 * ND_DECL( (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), int, 3, 100, 100, 100)
		 */
		SAC_ND_DECL__DATA ( (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), int, )
		SAC_ND_DECL__DESC ( (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), )
		const int SAC_ND_A_MIRROR_SHAPE ( (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 0) = 100;
		const int SAC_ND_A_MIRROR_SHAPE ( (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 1) = 100;
		const int SAC_ND_A_MIRROR_SHAPE ( (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 2) = 100;
		const int SAC_ND_A_MIRROR_SIZE ( (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))) = 1000000;
		const int SAC_ND_A_MIRROR_DIM ( (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))) = 3;

		/*
		 * ND_DECL( (SACp_emal_4718_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 3, 100, 100, 100)
		 */
		SAC_ND_DECL__DATA ( (SACp_emal_4718_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
		SAC_ND_DECL__DESC ( (SACp_emal_4718_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
		const int SAC_ND_A_MIRROR_SHAPE ( (SACp_emal_4718_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = 100;
		const int SAC_ND_A_MIRROR_SHAPE ( (SACp_emal_4718_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1) = 100;
		const int SAC_ND_A_MIRROR_SHAPE ( (SACp_emal_4718_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 2) = 100;
		const int SAC_ND_A_MIRROR_SIZE ( (SACp_emal_4718_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 1000000;
		const int SAC_ND_A_MIRROR_DIM ( (SACp_emal_4718_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 3;

		/*
		 * ND_DECL( (SACp_wlidx_4480_a, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
		 */
		SAC_ND_DECL__DATA ( (SACp_wlidx_4480_a, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
		SAC_ND_DECL__DESC ( (SACp_wlidx_4480_a, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
		SAC_NOTHING()

		/*
		 * ND_DECL( (SACp_eat_21, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
		 */
		SAC_ND_DECL__DATA ( (SACp_eat_21, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
		SAC_ND_DECL__DESC ( (SACp_eat_21, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
		SAC_NOTHING()

		/*
		 * ND_DECL( (SACp_eat_20, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
		 */
		SAC_ND_DECL__DATA ( (SACp_eat_20, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
		SAC_ND_DECL__DESC ( (SACp_eat_20, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
		SAC_NOTHING()

		/*
		 * ND_DECL( (SACp_eat_19, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
		 */
		SAC_ND_DECL__DATA ( (SACp_eat_19, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
		SAC_ND_DECL__DESC ( (SACp_eat_19, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
		SAC_NOTHING()


		SAC_INIT_LOCAL_MEM()
		SAC_CUDA_ALLOC_BEGIN ((SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 1, 3, int)
		/*
		 * ND_SET__SHAPE_arr( (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 3, 100, 100, 100)
		 */
		SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM ( (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))) == 3), 4, "Assignment with incompatible types found!");
		SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE ( (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 0) == 100), 4, "Assignment with incompatible types found!");
		SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE ( (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 1) == 100), 4, "Assignment with incompatible types found!");
		SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE ( (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 2) == 100), 4, "Assignment with incompatible types found!");
		SAC_NOOP()

		SAC_CUDA_ALLOC_END ((SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 1, 3, int)
		SAC_ND_ALLOC_BEGIN ((SACp_eat_21, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
		/*
		 * ND_SET__SHAPE_arr( (SACp_eat_21, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
		 */
		SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM ( (SACp_eat_21, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 4, "Assignment with incompatible types found!");
		SAC_NOOP()

		SAC_ND_ALLOC_END ((SACp_eat_21, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
		SAC_ND_ALLOC_BEGIN ((SACp_eat_20, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
		/*
		 * ND_SET__SHAPE_arr( (SACp_eat_20, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
		 */
		SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM ( (SACp_eat_20, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 4, "Assignment with incompatible types found!");
		SAC_NOOP()

		SAC_ND_ALLOC_END ((SACp_eat_20, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
		SAC_ND_ALLOC_BEGIN ((SACp_eat_19, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
		/*
		 * ND_SET__SHAPE_arr( (SACp_eat_19, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
		 */
		SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM ( (SACp_eat_19, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 4, "Assignment with incompatible types found!");
		SAC_NOOP()

		SAC_ND_ALLOC_END ((SACp_eat_19, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
		SAC_ND_ALLOC_BEGIN ((SACp_wlidx_4480_a, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
		/*
		 * ND_SET__SHAPE_arr( (SACp_wlidx_4480_a, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
		 */
		SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM ( (SACp_wlidx_4480_a, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 4, "Assignment with incompatible types found!");
		SAC_NOOP()

		SAC_ND_ALLOC_END ((SACp_wlidx_4480_a, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)

		SAC_GKCO_OPD_DECLARE (SAC_gkco_prt_5649_tmp)
		SAC_GKCO_OPD_DECLARE (SAC_gkco_prt_5650_ub)
		SAC_GKCO_OPD_REDEFINE (SAC_ND_READ ( (SACp_emal_4722__cnstass_4706_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), SAC_gkco_prt_5650_ub)


		SAC_GKCO_OPD_DECLARE (SAC_gkco_prt_5651_tmp)
		SAC_GKCO_OPD_DECLARE (SAC_gkco_prt_5652_lb)
		SAC_GKCO_OPD_DECLARE (SAC_gkco_prt_5653_ub)
		SAC_GKCO_OPD_DECLARE (SAC_gkco_prt_5654_st)
		SAC_GKCO_OPD_DECLARE (SAC_gkco_prt_5655_wi)
		SAC_GKCO_OPD_REDEFINE (SAC_gkco_prt_5653_ub, SAC_gkco_prt_5653_ub)

		SAC_GKCO_OPD_DECLARE (SAC_gkco_prt_5656_lb)
		SAC_GKCO_OPD_DECLARE (SAC_gkco_prt_5657_ub)
		SAC_GKCO_OPD_DECLARE (SAC_gkco_prt_5658_st)
		SAC_GKCO_OPD_DECLARE (SAC_gkco_prt_5659_wi)
		SAC_GKCO_OPD_REDEFINE (SAC_gkco_prt_5657_ub, SAC_gkco_prt_5657_ub)

		SAC_GKCO_OPD_DECLARE (SAC_gkco_prt_5660_lb)
		SAC_GKCO_OPD_DECLARE (SAC_gkco_prt_5661_ub)
		SAC_GKCO_OPD_DECLARE (SAC_gkco_prt_5662_st)
		SAC_GKCO_OPD_DECLARE (SAC_gkco_prt_5663_wi)
		SAC_GKCO_OPD_REDEFINE (SAC_gkco_prt_5661_ub, SAC_gkco_prt_5661_ub)


		SAC_GKCO_HOST_OPD_COMPRESS_SW (SAC_gkco_prt_5653_ub, SAC_gkco_prt_5653_ub, SAC_gkco_prt_5654_st, SAC_gkco_prt_5655_wi, SAC_gkco_prt_5651_tmp)

		SAC_GKCO_HOST_OPD_COMPRESS_SW (SAC_gkco_prt_5661_ub, SAC_gkco_prt_5661_ub, SAC_gkco_prt_5662_st, SAC_gkco_prt_5663_wi, SAC_gkco_prt_5651_tmp)


		SAC_GKCO_HOST_OPM_SET_GRID (2147483647, 65535, 65535, 0, SAC_gkco_prt_5653_ub, SAC_gkco_prt_5657_ub)

		SAC_GKCO_HOST_OPM_SET_BLOCK (1024, 1024, 64, 1024, SAC_gkco_prt_5661_ub)



		/*
		 * CUDA_GLOBALFUN_AP( SACf__MAIN___cuknl_5648_CUDA__i__i__i__i__i__i__i__id_100_100_100, 8, inout, int, 3, SAC_SET_NT_USG( FAG, (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))), in, int, 0, SAC_SET_NT_USG( FAG, (SACp_emal_4723__cnstass_4707_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), in, int, 0, SAC_SET_NT_USG( FAG, (SACp_emal_4722__cnstass_4706_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), in, int, 0, SAC_SET_NT_USG( FAG, (SACp_emal_4721__cnstass_4705_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), in, int, 0, SAC_SET_NT_USG( FAG, (SACp_emal_4726__cnstass_4710_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), in, int, 0, SAC_SET_NT_USG( FAG, (SACp_emal_4725__cnstass_4709_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), in, int, 0, SAC_SET_NT_USG( FAG, (SACp_emal_4724__cnstass_4708_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), in, int, 0, SAC_SET_NT_USG( FAG, (SACp_emal_4727__flat_13, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))))
		 */
		SAC_TR_GPU_PRINT ("   kernel name \"SACf__MAIN___cuknl_5648_CUDA__i__i__i__i__i__i__i__id_100_100_100\"\n");
		SAC_PF_BEGIN_CUDA_KNL ();
		SACf__MAIN___cuknl_5648_CUDA__i__i__i__i__i__i__i__id_100_100_100 <<< grid, block>>> (    SAC_CUDA_ARG_inout ( SAC_SET_NT_USG ( FAG, (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))), int), SAC_ND_A_MIRROR_SHAPE (SAC_SET_NT_USG ( FAG, (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))), 0), SAC_ND_A_MIRROR_SHAPE (SAC_SET_NT_USG ( FAG, (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))), 1), SAC_ND_A_MIRROR_SHAPE (SAC_SET_NT_USG ( FAG, (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))), 2), SAC_ND_A_MIRROR_SIZE (SAC_SET_NT_USG ( FAG, (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))))), SAC_ND_A_MIRROR_DIM (SAC_SET_NT_USG ( FAG, (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))))),     SAC_CUDA_ARG_in ( SAC_SET_NT_USG ( FAG, (SACp_emal_4723__cnstass_4707_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int),     SAC_CUDA_ARG_in ( SAC_SET_NT_USG ( FAG, (SACp_emal_4722__cnstass_4706_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int),     SAC_CUDA_ARG_in ( SAC_SET_NT_USG ( FAG, (SACp_emal_4721__cnstass_4705_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int),     SAC_CUDA_ARG_in ( SAC_SET_NT_USG ( FAG, (SACp_emal_4726__cnstass_4710_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int),     SAC_CUDA_ARG_in ( SAC_SET_NT_USG ( FAG, (SACp_emal_4725__cnstass_4709_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int),     SAC_CUDA_ARG_in ( SAC_SET_NT_USG ( FAG, (SACp_emal_4724__cnstass_4708_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int),     SAC_CUDA_ARG_in ( SAC_SET_NT_USG ( FAG, (SACp_emal_4727__flat_13, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int));
		SAC_PF_END_CUDA_KNL ();
		SAC_CUDA_GET_LAST_KERNEL_ERROR();
	}

	/*
	 * ND_REFRESH__MIRROR( (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 3)
	 */
	SAC_NOOP()

	SAC_ND_FREE ((SACp_emal_4723__cnstass_4707_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
	SAC_ND_FREE ((SACp_emal_4722__cnstass_4706_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
	SAC_ND_FREE ((SACp_emal_4721__cnstass_4705_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
	SAC_ND_FREE ((SACp_emal_4726__cnstass_4710_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
	SAC_ND_FREE ((SACp_emal_4725__cnstass_4709_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
	SAC_ND_FREE ((SACp_emal_4724__cnstass_4708_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
	SAC_ND_FREE ((SACp_eat_19, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
	SAC_ND_FREE ((SACp_eat_20, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
	SAC_ND_FREE ((SACp_eat_21, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
	SAC_ND_FREE ((SACp_emal_4727__flat_13, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
	SAC_ND_FREE ((SACp_wlidx_4480_a, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
	SAC_ND_ALLOC_BEGIN ((SACp_emal_4718_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 3, int)
	/*
	 * ND_SET__SHAPE_arr( (SACp_emal_4718_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3, 100, 100, 100)
	 */
	SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM ( (SACp_emal_4718_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 3), 4, "Assignment with incompatible types found!");
	SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE ( (SACp_emal_4718_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) == 100), 4, "Assignment with incompatible types found!");
	SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE ( (SACp_emal_4718_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1) == 100), 4, "Assignment with incompatible types found!");
	SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE ( (SACp_emal_4718_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 2) == 100), 4, "Assignment with incompatible types found!");
	SAC_NOOP()

	SAC_ND_ALLOC_END ((SACp_emal_4718_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 3, int)
	/*
	 * CUDA_MEM_TRANSFER( (SACp_emal_4718_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), int, hipMemcpyDeviceToHost)
	 */
	SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SIZE ( (SACp_emal_4718_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == SAC_ND_A_SIZE ( (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))))), 4, "hipMemcpy: Destionation and source arrays should have equal sizes!");
	SAC_TR_GPU_PRINT ("hipMemcpyDeviceToHost size %d (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))) -> (SACp_emal_4718_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))\n", SAC_ND_A_SIZE ( (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))));
	SAC_PF_BEGIN_CUDA_DtoH()
	SAC_CUDA_MEM_TRANSFER ((SACp_emal_4718_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), int, hipMemcpyDeviceToHost)
	SAC_PF_END_CUDA_DtoH()

	SAC_CUDA_FREE ((SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), )
	/*
	 * ND_FUN_RET( , 1, out, (SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), (SACp_emal_4718_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
	 */
	SAC_ND_RET_out ( (SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), (SACp_emal_4718_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
	return;
	SAC_CLEANUP_LOCAL_MEM()
}
/*
   * ND_FUN_DEF_END( SACf__MAIN__hiding, , 1, out, int, (SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * WITH-loop Count: 0
 * _MAIN::SACf__MAIN__main(...) [ body ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACf__MAIN__main, , 1, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DEF_FUN_BEGIN2 ( SACf__MAIN__main, void,  SAC_ND_PARAM_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int)) {
	SAC_HM_DEFINE_THREAD_STATUS ( SAC_HM_single_threaded)
	SAC_MT_DEFINE_ST_SELF()

	{
		SAC_ND_DECL_CONST__DATA ((SACp_emal_4729__ivesplit_373, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 232221)
		/*
		 * ND_DECL( (SACp_emal_4728__flat_14, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
		 */
		SAC_ND_DECL__DATA ( (SACp_emal_4728__flat_14, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
		SAC_ND_DECL__DESC ( (SACp_emal_4728__flat_14, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
		SAC_NOTHING()

		/*
		 * ND_DECL( (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 3, 100, 100, 100)
		 */
		SAC_ND_DECL__DATA ( (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
		SAC_ND_DECL__DESC ( (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
		const int SAC_ND_A_MIRROR_SHAPE ( (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = 100;
		const int SAC_ND_A_MIRROR_SHAPE ( (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1) = 100;
		const int SAC_ND_A_MIRROR_SHAPE ( (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 2) = 100;
		const int SAC_ND_A_MIRROR_SIZE ( (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 1000000;
		const int SAC_ND_A_MIRROR_DIM ( (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 3;


		SAC_INIT_LOCAL_MEM()
		/*
		 * ND_FUN_AP( SACf__MAIN__hiding, , 1, out, int, SAC_SET_NT_USG( FAG, (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))))
		 */
		SAC_ND_FUNAP2 ( SACf__MAIN__hiding,  SAC_ND_ARG_out ( SAC_SET_NT_USG ( FAG, (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int))

		/*
		 * ND_REFRESH__MIRROR( (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3)
		 */
		SAC_NOOP()

		/*
		 * ND_PRF_IDX_SEL__DATA( (SACp_emal_4728__flat_14, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3, (SACp_emal_4729__ivesplit_373, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
		 */
		SAC_TR_PRF_PRINT ( ("ND_PRF_IDX_SEL__DATA( (SACp_emal_4728__flat_14, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3, (SACp_emal_4729__ivesplit_373, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))"))
		SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM ( (SACp_emal_4729__ivesplit_373, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 16, "1st argument of _idx_sel_ is not a scalar!");
		SAC_ND_WRITE_READ_COPY ( (SACp_emal_4728__flat_14, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), SAC_ND_READ ( (SACp_emal_4729__ivesplit_373, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), )

		SAC_ND_FREE ((SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
		SAC_ND_FREE ((SACp_emal_4729__ivesplit_373, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
		/*
		 * ND_FUN_RET( , 1, out, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), (SACp_emal_4728__flat_14, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
		 */
		SAC_ND_RET_out ( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), (SACp_emal_4728__flat_14, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
		return;
		SAC_CLEANUP_LOCAL_MEM()
	}
	/*
	   * ND_FUN_DEF_END( SACf__MAIN__main, , 1, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
	   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * WITH-loop Count: 0
 * _MAIN::SACf__MAIN___cuknl_5648_CUDA__i__i__i__i__i__i__i__id_100_100_100(...) [ body ]
 ****************************************************************************/
/*
 * CUDA_GLOBALFUN_DEF_BEGIN( SACf__MAIN___cuknl_5648_CUDA__i__i__i__i__i__i__i__id_100_100_100, 8, inout, int, (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 3, in, int, (SACp_ub_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_ub_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_ub_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_emal_4727__flat_13, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0)
 */
__global__ void SACf__MAIN___cuknl_5648_CUDA__i__i__i__i__i__i__i__id_100_100_100 (SAC_CUDA_PARAM_inout ( (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), int), int SAC_ND_A_MIRROR_SHAPE ((SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 0), int SAC_ND_A_MIRROR_SHAPE ((SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 1), int SAC_ND_A_MIRROR_SHAPE ((SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 2), int SAC_ND_A_MIRROR_SIZE ((SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, ))))))))))), int SAC_ND_A_MIRROR_DIM ((SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, ))))))))))), SAC_CUDA_PARAM_in ( (SACp_ub_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in ( (SACp_ub_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in ( (SACp_ub_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in ( (SACp_lb_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in ( (SACp_lb_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in ( (SACp_lb_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in ( (SACp_emal_4727__flat_13, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int)) {

	{
		/*
		 * ND_DECL( (SACp_wlidx_4480_a, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
		 */
		SAC_ND_DECL__DATA ( (SACp_wlidx_4480_a, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
		SAC_ND_DECL__DESC ( (SACp_wlidx_4480_a, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
		SAC_NOTHING()

		/*
		 * ND_DECL( (SACp_eat_19, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
		 */
		SAC_ND_DECL__DATA ( (SACp_eat_19, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
		SAC_ND_DECL__DESC ( (SACp_eat_19, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
		SAC_NOTHING()

		/*
		 * ND_DECL( (SACp_eat_20, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
		 */
		SAC_ND_DECL__DATA ( (SACp_eat_20, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
		SAC_ND_DECL__DESC ( (SACp_eat_20, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
		SAC_NOTHING()

		/*
		 * ND_DECL( (SACp_eat_21, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
		 */
		SAC_ND_DECL__DATA ( (SACp_eat_21, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
		SAC_ND_DECL__DESC ( (SACp_eat_21, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
		SAC_NOTHING()

		/*
		 * CUDA_DECL_KERNEL_ARRAY( (SACl_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 1, 3)
		 */
		int SAC_ND_A_FIELD ( (SACl_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))[3];
		SAC_ND_DECL__DESC ( (SACl_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
		const int SAC_ND_A_MIRROR_SHAPE ( (SACl_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = 3;
		const int SAC_ND_A_MIRROR_SIZE ( (SACl_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 3;
		const int SAC_ND_A_MIRROR_DIM ( (SACl_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 1;


		SAC_INIT_LOCAL_MEM()
		/*
		 * CUDA_WLIDS( (SACp_eat_19, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, 3, 2, (SACl_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), false)
		 */
		SAC_CUDA_WLIDS_HD ( (SACp_eat_19, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, THREADIDX_X, SACp_step_2, SACp_width_2, SACp_lb_2, SACp_ub_2)
		SAC_ND_WRITE ( (SACl_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 2) = SAC_ND_READ ( (SACp_eat_19, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0);

		/*
		 * CUDA_WLIDS( (SACp_eat_20, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, 3, 1, (SACl_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), false)
		 */
		SAC_CUDA_WLIDS_HD ( (SACp_eat_20, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, BLOCKIDX_X, SACp_step_1, SACp_width_1, SACp_lb_1, SACp_ub_1)
		SAC_ND_WRITE ( (SACl_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1) = SAC_ND_READ ( (SACp_eat_20, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0);

		/*
		 * CUDA_WLIDS( (SACp_eat_21, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, 3, 0, (SACl_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), false)
		 */
		SAC_CUDA_WLIDS_HD ( (SACp_eat_21, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, BLOCKIDX_Y, SACp_step_0, SACp_width_0, SACp_lb_0, SACp_ub_0)
		SAC_ND_WRITE ( (SACl_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = SAC_ND_READ ( (SACp_eat_21, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0);

		/*
		 * ND_IDXS2OFFSET_arr( (SACp_wlidx_4480_a, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3, (SACp_eat_21, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_eat_20, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_eat_19, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3, 100, 100, 100)
		 */
		SAC_ND_WRITE ( (SACp_wlidx_4480_a, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = ( 100 * ( 100 * SAC_ND_READ ( (SACp_eat_21, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) + SAC_ND_READ ( (SACp_eat_20, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) ) + SAC_ND_READ ( (SACp_eat_19, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) );

		/*
		 * CUDA_WL_ASSIGN( (SACp_emal_4727__flat_13, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 3, (SACp_wlidx_4480_a, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
		 */
		SAC_ND_WRITE_READ_COPY ( (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SAC_ND_READ ( (SACp_wlidx_4480_a, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), (SACp_emal_4727__flat_13, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, );

		SAC_ND_FREE ((SACp_wlidx_4480_a, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
		SAC_ND_FREE ((SACp_eat_19, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
		SAC_ND_FREE ((SACp_eat_20, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
		SAC_ND_FREE ((SACp_eat_21, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
		SAC_NOOP()

		SAC_CLEANUP_LOCAL_MEM()
	}
	/*
	 * CUDA_GLOBALFUN_DEF_END( SACf__MAIN___cuknl_5648_CUDA__i__i__i__i__i__i__i__id_100_100_100, 8, inout, int, (SACp_emal_4719__iwlmem_4704_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 3, in, int, (SACp_ub_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_ub_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_ub_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_emal_4727__flat_13, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0)
	 */
}

/*
 * stubs for SACARGfreeDataUdt and SACARGcopyDataUdt
 */
extern "C" void SACARGfreeDataUdt ( int, void *);
extern "C" void *SACARGcopyDataUdt ( int, int, void *);
void SACARGfreeDataUdt ( int size, void *data) {}
void *SACARGcopyDataUdt ( int type, int size, void *data) {
	return ((void *) 0x0);
}

int main ( int __argc, char *__argv[]) {
	SAC_ND_DECL__DATA ( (SAC_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
	SAC_ND_DECL__DESC ( (SAC_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
	SAC_NOTHING()
	SAC_HWLOC_SETUP();
	SAC_MT_SETUP_INITIAL();
	SAC_PF_SETUP();
	SAC_HM_SETUP();
	SAC_MT_SETUP();
	SAC_CUDA_SETUP();
	SAC_CS_SETUP();
	SAC_COMMANDLINE_SET ( __argc, __argv);

	SAC_INVOKE_MAIN_FUN ( SACf__MAIN__main, SAC_ND_ARG_out ( (SAC_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int));

	SAC_DISTMEM_BARRIER();
	SAC_PF_PRINT();
	SAC_CS_FINALIZE();
	SAC_MT_FINALIZE();
	SAC_CUDA_FINALIZE();
	SAC_HWLOC_FINALIZE();
	SAC_HM_PRINT();

	return ( SAC_ND_READ ( (SAC_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0));
}
